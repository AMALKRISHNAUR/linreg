#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            fprintf(stderr, "cuBLAS error in %s:%d: %d\n", \
                    __FILE__, __LINE__, status); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define TILE_SIZE 16

// Tiled matrix multiplication kernel
__global__ void tiledMatrixMulKernel(float *A, float *B, float *C, int m, int n, int p) {
    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; t++) {
        int a_col = t * TILE_SIZE + threadIdx.x;
        if (row < m && a_col < n) {
            s_A[threadIdx.y][threadIdx.x] = A[row * n + a_col];
        } else {
            s_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        int b_row = t * TILE_SIZE + threadIdx.y;
        if (col < p && b_row < n) {
            s_B[threadIdx.y][threadIdx.x] = B[b_row * p + col];
        } else {
            s_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        if (row < m && col < p) {
            for (int k = 0; k < TILE_SIZE && k < n; k++) {
                sum += s_A[threadIdx.y][k] * s_B[k][threadIdx.x];
            }
        }

        __syncthreads();
    }

    if (row < m && col < p) {
        C[row * p + col] = sum;
    }
}

// Kernel to compute X^T y
__global__ void computeXTyKernel(float *X, float *y, float *XTy, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < cols) {
        float sum = 0.0f;
        for (int i = 0; i < rows; i++) {
            sum += X[i * cols + col] * y[i];
        }
        XTy[col] = sum;
    }
}

// Read X
void readX(const char *filename, float *X, int rows, int cols, int features) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file %s\n", filename);
        exit(1);
    }
    char line[1024];
    fgets(line, 1024, file);
    for (int i = 0; i < rows; i++) {
        if (fscanf(file, "%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f",
                   &X[i * cols + 0], &X[i * cols + 1], &X[i * cols + 2], &X[i * cols + 3],
                   &X[i * cols + 4], &X[i * cols + 5], &X[i * cols + 6], &X[i * cols + 7],
                   &X[i * cols + 8], &X[i * cols + 9], &X[i * cols + 10], &X[i * cols + 11],
                   &X[i * cols + 12]) != features) {
            fprintf(stderr, "Error reading row %d in %s\n", i, filename);
            fclose(file);
            exit(1);
        }
    }
    fclose(file);
    for (int i = 0; i < rows; i++) {
        X[i * cols + cols - 1] = 1.0f;
    }
    printf("Sample X[0]: ");
    for (int j = 0; j < cols; j++) {
        printf("%.4f ", X[j]);
    }
    printf("\n");
}

// Read y
void readY(const char *filename, float *y, int rows) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        fprintf(stderr, "Error opening file %s\n", filename);
        exit(1);
    }
    char line[1024];
    fgets(line, 1024, file);
    for (int i = 0; i < rows; i++) {
        if (fscanf(file, "%f", &y[i]) != 1) {
            fprintf(stderr, "Error reading row %d in %s\n", i, filename);
            fclose(file);
            exit(1);
        }
    }
    fclose(file);
    printf("Sample y[0]: %.4f\n", y[0]);
}

int main() {
    const int rows = 256;
    const int features = 13;
    const int cols = features + 1;
    const char *x_filename = "X_train.csv";
    const char *y_filename = "y_train.csv";

    // Host memory
    float *h_X = (float *)malloc(rows * cols * sizeof(float));
    float *h_y = (float *)malloc(rows * sizeof(float));
    float *h_beta = (float *)calloc(cols, sizeof(float));
    float *h_XTX = (float *)malloc(cols * cols * sizeof(float));

    if (!h_X || !h_y || !h_beta || !h_XTX) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(1);
    }

    for (int i = 0; i < rows * cols; i++) {
        h_X[i] = 0.0f;
    }

    // Read datasets
    printf("Reading X from %s\n", x_filename);
    readX(x_filename, h_X, rows, cols, features);
    printf("Reading y from %s\n", y_filename);
    readY(y_filename, h_y, rows);

    // Device memory
    float *d_X, *d_y, *d_XTX, *d_XTX_inv, *d_XTy, *d_beta;
    float **d_XTX_array, **d_XTX_inv_array;
    CUDA_CHECK(hipMalloc(&d_X, rows * cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_y, rows * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_XTX, cols * cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_XTX_inv, cols * cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_XTy, cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_beta, cols * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_XTX_array, sizeof(float *)));
    CUDA_CHECK(hipMalloc(&d_XTX_inv_array, sizeof(float *)));

    // Set pointer arrays
    CUDA_CHECK(hipMemcpy(d_XTX_array, &d_XTX, sizeof(float *), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_XTX_inv_array, &d_XTX_inv, sizeof(float *), hipMemcpyHostToDevice));

    // Copy data
    printf("Copying X to device\n");
    CUDA_CHECK(hipMemcpy(d_X, h_X, rows * cols * sizeof(float), hipMemcpyHostToDevice));
    printf("Copying y to device\n");
    CUDA_CHECK(hipMemcpy(d_y, h_y, rows * sizeof(float), hipMemcpyHostToDevice));

    // Initialize cuBLAS
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Compute X^T X
    printf("Computing X^T X\n");
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((cols + TILE_SIZE - 1) / TILE_SIZE, (cols + TILE_SIZE - 1) / TILE_SIZE);
    tiledMatrixMulKernel<<<gridSize, blockSize>>>(d_X, d_X, d_XTX, cols, rows, cols);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Debug: Copy X^T X back
    CUDA_CHECK(hipMemcpy(h_XTX, d_XTX, cols * cols * sizeof(float), hipMemcpyDeviceToHost));
    printf("Sample X^T X[0][0..3]: %.4f %.4f %.4f %.4f\n",
           h_XTX[0], h_XTX[1], h_XTX[2], h_XTX[3]);

    // Compute X^T y
    printf("Computing X^T y\n");
    blockSize = dim3(256, 1);
    gridSize = dim3((cols + blockSize.x - 1) / blockSize.x, 1);
    computeXTyKernel<<<gridSize, blockSize>>>(d_X, d_y, d_XTy, rows, cols);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Matrix inversion
    printf("Inverting X^T X\n");
    int *d_pivotArray, *d_infoArray;
    CUDA_CHECK(hipMalloc(&d_pivotArray, cols * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_infoArray, sizeof(int)));

    CUBLAS_CHECK(hipblasSgetrfBatched(handle, cols, d_XTX_array, cols, d_pivotArray, d_infoArray, 1));
    CUDA_CHECK(hipDeviceSynchronize());

    int h_info;
    CUDA_CHECK(hipMemcpy(&h_info, d_infoArray, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        fprintf(stderr, "LU decomposition failed: info=%d\n", h_info);
        exit(1);
    }

    CUBLAS_CHECK(hipblasSgetriBatched(handle, cols, (const float **)d_XTX_array, cols, d_pivotArray,
                                     d_XTX_inv_array, cols, d_infoArray, 1));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(&h_info, d_infoArray, sizeof(int), hipMemcpyDeviceToHost));
    if (h_info != 0) {
        fprintf(stderr, "Matrix inversion failed: info=%d\n", h_info);
        exit(1);
    }

    // Compute beta
    printf("Computing beta\n");
    blockSize = dim3(TILE_SIZE, TILE_SIZE);
    gridSize = dim3(1, cols / TILE_SIZE + 1);
    tiledMatrixMulKernel<<<gridSize, blockSize>>>(d_XTX_inv, d_XTy, d_beta, cols, cols, 1);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy beta
    printf("Copying beta to host\n");
    CUDA_CHECK(hipMemcpy(h_beta, d_beta, cols * sizeof(float), hipMemcpyDeviceToHost));

    // Print coefficients
    printf("Linear Regression Coefficients:\n");
    for (int i = 0; i < features; i++) {
        printf("Weight w%d (for feature %d) = %.4f\n", i + 1, i + 1, h_beta[i]);
    }
    printf("Bias b = %.4f\n", h_beta[cols - 1]);

    // Cleanup
    free(h_X); free(h_y); free(h_beta); free(h_XTX);
    CUDA_CHECK(hipFree(d_X)); CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(d_XTX)); CUDA_CHECK(hipFree(d_XTX_inv));
    CUDA_CHECK(hipFree(d_XTy)); CUDA_CHECK(hipFree(d_beta));
    CUDA_CHECK(hipFree(d_XTX_array)); CUDA_CHECK(hipFree(d_XTX_inv_array));
    CUDA_CHECK(hipFree(d_pivotArray)); CUDA_CHECK(hipFree(d_infoArray));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return 0;
}